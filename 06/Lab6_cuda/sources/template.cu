#include "hip/hip_runtime.h"
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}

#include <gputk.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define gpuTKCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      gpuTKLog(ERROR, "Failed to run stmt ", #stmt);                         \
      gpuTKLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// __global__ void scan_add(float *output, float *auxOutput, int lenOutput, int lenAuxOutput)
// {
//   if (blockIdx.x > 0 && blockIdx.x < lenAuxOutput - 1)
//   {
//     int idx = SECTION_SIZE * blockIdx.x + threadIdx.x;
//     output[idx] += auxOutput[blockIdx.x - 1];
//     output[idx + 1] += auxOutput[blockIdx.x - 1];
//   }
// }

__global__ void scan(float *input, float *output, int len) {
  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from here

  __shared__ float T[BLOCK_SIZE];
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < len)
  {
    T[threadIdx.x] = input[idx];
  }
  else
  {
    T[threadIdx.x] = 0;
  }
  __syncthreads();

  for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2)
  {
    __syncthreads();

    int index = (threadIdx.x + 1) * 2 * stride - 1;

    if (index < blockDim.x)
    {
      T[index] += T[index - stride];
    }
  }

  // postscan
  for (unsigned int stride = blockDim.x / 4; stride > 0; stride /= 2)
  {
    __syncthreads();

    int index = (threadIdx.x + 1) * 2 * stride - 1;
    if (index + stride < blockDim.x)
    {
      T[index + stride] += T[index];
    }
  }

  // put back to output array
  if (idx < len)
  {
    output[idx] = T[threadIdx.x];
  }
}

int main(int argc, char **argv) {
  gpuTKArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numElements; // number of elements in the list

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The number of input elements in the input is ",
        numElements);

  gpuTKTime_start(GPU, "Allocating GPU memory.");
  gpuTKCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  gpuTKCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Clearing output memory.");
  gpuTKCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  gpuTKTime_stop(GPU, "Clearing output memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  gpuTKCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));
  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 blockDim(BLOCK_SIZE, 1, 1);
  dim3 gridDim((numElements - 1) / BLOCK_SIZE + 1, 1, 1);

  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  scan<<<gridDim, blockDim>>>(deviceInput, deviceOutput, numElements);

  if (numElements > BLOCK_SIZE)
  {
    // int numAuxArray = numElements / SECTION_SIZE;
    // float *deviceAuxInput, *deviceAuxOutput;
    // hipMalloc(&deviceAuxInput, numAuxArray * sizeof(float));
    // hipMalloc(&deviceAuxOutput, numAuxArray * sizeof(float));
    // for (int i = 1; i < numAuxArray; i++)
    // {
    //   deviceAuxInput[i] = deviceOutput[i * SECTION_SIZE - 1];
    // }
    // dim3 gridDim2((numAuxArray - 1) / SECTION_SIZE + 1, 1, 1);
    // scan<<<gridDim2, blockDim>>>(deviceAuxInput, deviceAuxOutput, numAuxArray);

    // // add scanned block sum i
    // scan_add<<<gridDim, blockDim>>>(deviceOutput, deviceAuxOutput, numElements, numAuxArray);
  }

  hipDeviceSynchronize();
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  gpuTKCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                     hipMemcpyDeviceToHost));
  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  gpuTKSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}
