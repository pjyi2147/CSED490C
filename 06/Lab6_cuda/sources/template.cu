#include "hip/hip_runtime.h"
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}

#include <gputk.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define gpuTKCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      gpuTKLog(ERROR, "Failed to run stmt ", #stmt);                         \
      gpuTKLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void scan(float *input, float *output, int len) {
  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from here
}

int main(int argc, char **argv) {
  gpuTKArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numElements; // number of elements in the list

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The number of input elements in the input is ",
        numElements);

  gpuTKTime_start(GPU, "Allocating GPU memory.");
  gpuTKCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  gpuTKCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Clearing output memory.");
  gpuTKCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  gpuTKTime_stop(GPU, "Clearing output memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  gpuTKCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));
  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here

  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce

  hipDeviceSynchronize();
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  gpuTKCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                     hipMemcpyDeviceToHost));
  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  gpuTKSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}
