#include "hip/hip_runtime.h"
#include <gputk.h>

#define NUM_BINS 4096

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

void histogram(unsigned int *input, unsigned int *bins,
               unsigned int num_elements, unsigned int num_bins) {
  __shared__ unsigned int private_histo[NUM_BINS];
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  while (i < num_elements) {
    atomicAdd(&(private_histo[input[i]]), 1);
    i += stride;
  }
  __syncthreads();
}

int main(int argc, char *argv[]) {
  gpuTKArg_t args;
  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (unsigned int *)gpuTKImport(gpuTKArg_getInputFile(args, 0),
                                       &inputLength, "Integer");
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The input length is ", inputLength);
  gpuTKLog(TRACE, "The number of bins is ", NUM_BINS);

  gpuTKTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceInput, inputLength * sizeof(unsigned int));
  hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(unsigned int));
  CUDA_CHECK(hipDeviceSynchronize());
  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, inputLength * sizeof(unsigned int),
             hipMemcpyHostToDevice);
  CUDA_CHECK(hipDeviceSynchronize());
  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  // Launch kernel
  // ----------------------------------------------------------
  gpuTKLog(TRACE, "Launching kernel");
  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Perform kernel computation here
  dim3 dimBlock(1024, 1, 1);
  dim3 dimGrid((inputLength - 1) / 1024 + 1, 1, 1);
  histogram<<<dimGrid, dimBlock>>>(deviceInput, deviceBins, inputLength, NUM_BINS);
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(unsigned int),
             hipMemcpyDeviceToHost);
  CUDA_CHECK(hipDeviceSynchronize());
  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceBins);
  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  // Verify correctness
  // -----------------------------------------------------
  gpuTKSolution(args, hostBins, NUM_BINS);

  free(hostBins);
  free(hostInput);
  return 0;
}
