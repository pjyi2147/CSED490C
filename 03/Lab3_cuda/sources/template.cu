#include "hip/hip_runtime.h"
#include <gputk.h>


__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < len) {
    out[index] = in1[index] + in2[index];
  }
}

int main(int argc, char **argv) {
  gpuTKArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;
  unsigned int numStreams;

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)gpuTKImport(gpuTKArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The input length is ", inputLength);

  gpuTKTime_start(GPU, "Allocating Pinned memory.");

  //@@ Allocate GPU memory here using pinned memory here

  //@@ Create and setup streams 
  //@@ Calculate data segment size of input data processed by each stream 

 
  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Perform parallel vector addition with different streams. 
  for (unsigned int s = 0; s<numStreams; s++){
          //@@ Asynchronous copy data to the device memory in segments 
          //@@ Calculate starting and ending indices for per-stream data

          //@@ Invoke CUDA Kernel
          //@@ Determine grid and thread block sizes (consider ococupancy)     

          //@@ Asynchronous copy data from the device memory in segments 

  }

  //@@ Synchronize

  gpuTKTime_stop(Compute, "Performing CUDA computation");


  gpuTKTime_start(GPU, "Freeing Pinned Memory");
  //@@ Destory cudaStream


  //@@ Free the GPU memory here


  gpuTKTime_stop(GPU, "Freeing Pinned Memory");

  gpuTKSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
